#include "hip/hip_runtime.h"
// #include <hip/hip_vector_types.h>
#ifdef _WIN32
#include <windows.h>
#endif
#include "algorithmparameters.h"
#include "cameraparameters.h"
#include "config.h"
#include "globalstate.h"
#include "imageinfo.h"
#include "linestate.h"
#include <stdint.h>  // for uint8_t
#include <stdio.h>

#include "hip/hip_runtime_api.h"
#include "vector_operations.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <math.h>
#include <string>
#include <vector>
#include <hip/hip_vector_types.h>  // float4

using namespace std;

#ifndef SHARED_HARDCODED
__managed__ int SHARED_SIZE_W_m;
__constant__ int SHARED_SIZE_W;
__managed__ int SHARED_SIZE_H;
__managed__ int SHARED_SIZE = 0;
__managed__ int WIN_RADIUS_W;
__managed__ int WIN_RADIUS_H;
__managed__ int TILE_W;
__managed__ int TILE_H;
#endif

__device__ FORCEINLINE_GIPUMA float curand_between(hiprandState *cs,
                                                   const float &min,
                                                   const float &max) {
    return hiprand_uniform(cs) * (max - min) + min;
}

__device__ FORCEINLINE_GIPUMA static void rndUnitVectorSphereMarsaglia_cu(
    float3 *v, hiprandState *cs) {
    float x = 1.0f;
    float y = 1.0f;
    float sum = 2.0f;
    while (sum >= 1.0f) {
        x = curand_between(cs, -1.0f, 1.0f);
        y = curand_between(cs, -1.0f, 1.0f);
        sum = get_pow2_norm(x, y);
    }
    const float sq = sqrtf(1.0f - sum);
    v->x = 2.0f * x * sq;
    v->y = 2.0f * y * sq;
    v->z = 1.0f - 2.0f * sum;
}

template <typename T>
__global__ void gipuma_init_cu2(GlobalState &gs) {
    const int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                             blockIdx.y * blockDim.y + threadIdx.y);
    const int rows = gs.cameras->rows;
    const int cols = gs.cameras->cols;
    if (p.x >= cols) return;
    if (p.y >= rows) return;
    // printf("gipuma_init_cu2 x, y: %d %d \n", p.x, p.y);

    const int center = p.y * cols + p.x;
    hiprandState localState = gs.cs[center];
    hiprand_init(clock64(), p.y, p.x, &localState);

    float3 generatedUnitDir;
    rndUnitVectorSphereMarsaglia_cu(&generatedUnitDir, &localState);
    gs.lines->unitDirection[center] = generatedUnitDir;

    // use disparity instead of depth?
    float mind = gs.params->depthMin;
    float maxd = gs.params->depthMax;
    gs.lines->depth[center] = curand_between(&localState, mind, maxd);

    // TODO: compute and save cost
    return;
}

template <typename T>
__global__ void gipuma_black_spatialPropClose_cu(GlobalState &gs) {
    int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                       blockIdx.y * blockDim.y + threadIdx.y);
    if (threadIdx.x % 2 == 0)
        p.y = p.y * 2;
    else
        p.y = p.y * 2 + 1;
    printf("gipuma_black_spatialPropClose_cu x, y: %d %d \n", p.x, p.y);
    int2 tile_offset;
    tile_offset.x = blockIdx.x * blockDim.x - WIN_RADIUS_W;
    tile_offset.y = 2.0 * blockIdx.y * blockDim.y - WIN_RADIUS_H;
    // gipuma_checkerboard_spatialPropClose_cu<T>(gs, p, tile_offset, iter);
}

template <typename T>
__global__ void gipuma_black_spatialPropFar_cu(GlobalState &gs) {
    int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                       blockIdx.y * blockDim.y + threadIdx.y);
    if (threadIdx.x % 2 == 0)
        p.y = p.y * 2;
    else
        p.y = p.y * 2 + 1;
    int2 tile_offset;
    tile_offset.x = blockIdx.x * blockDim.x - WIN_RADIUS_W;
    tile_offset.y = 2.0 * blockIdx.y * blockDim.y - WIN_RADIUS_H;
    printf("gipuma_black_spatialPropFar_cu x, y: %d %d \n", p.x, p.y);
    // gipuma_checkerboard_spatialPropFar_cu<T>(gs, p, tile_offset, iter);
}

template <typename T>
__global__ void gipuma_black_lineRefine_cu(GlobalState &gs) {
    int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                       blockIdx.y * blockDim.y + threadIdx.y);
    if (threadIdx.x % 2 == 0)
        p.y = p.y * 2;
    else
        p.y = p.y * 2 + 1;
    int2 tile_offset;
    tile_offset.x = blockIdx.x * blockDim.x - WIN_RADIUS_W;
    tile_offset.y = 2.0 * blockIdx.y * blockDim.y - WIN_RADIUS_H;
    printf("gipuma_black_lineRefine_cu x, y: %d %d \n", p.x, p.y);

    // gipuma_checkerboard_lineRefinement_cu<T>(gs, p, tile_offset, iter);
}

template <typename T>
__global__ void gipuma_red_spatialPropClose_cu(GlobalState &gs) {
    int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                       blockIdx.y * blockDim.y + threadIdx.y);
    if (threadIdx.x % 2 == 0)
        p.y = p.y * 2 + 1;
    else
        p.y = p.y * 2;
    int2 tile_offset;
    tile_offset.x = blockIdx.x * blockDim.x - WIN_RADIUS_W;
    tile_offset.y = 2.0 * blockIdx.y * blockDim.y - WIN_RADIUS_H;
    printf("gipuma_red_spatialPropClose_cu x, y: %d %d \n", p.x, p.y);

    // gipuma_checkerboard_spatialPropClose_cu<T>(gs, p, tile_offset, iter);
}

template <typename T>
__global__ void gipuma_red_spatialPropFar_cu(GlobalState &gs) {
    int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                       blockIdx.y * blockDim.y + threadIdx.y);
    if (threadIdx.x % 2 == 0)
        p.y = p.y * 2 + 1;
    else
        p.y = p.y * 2;
    int2 tile_offset;
    tile_offset.x = blockIdx.x * blockDim.x - WIN_RADIUS_W;
    tile_offset.y = 2.0 * blockIdx.y * blockDim.y - WIN_RADIUS_H;
    printf("gipuma_red_spatialPropFar_cu x, y: %d %d \n", p.x, p.y);

    // gipuma_checkerboard_spatialPropFar_cu<T>(gs, p, tile_offset, iter);
}

template <typename T>
__global__ void gipuma_red_lineRefine_cu(GlobalState &gs) {
    int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                       blockIdx.y * blockDim.y + threadIdx.y);
    if (threadIdx.x % 2 == 0)
        p.y = p.y * 2 + 1;
    else
        p.y = p.y * 2;
    int2 tile_offset;
    tile_offset.x = blockIdx.x * blockDim.x - WIN_RADIUS_W;
    tile_offset.y = 2.0 * blockIdx.y * blockDim.y - WIN_RADIUS_H;
    printf("gipuma_red_lineRefine_cu x, y: %d %d \n", p.x, p.y);

    // gipuma_checkerboard_lineRefinement_cu<T>(gs, p, tile_offset, iter);
}

template <typename T>
void gipuma(GlobalState &gs) {
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    int rows = gs.cameras->rows;
    int cols = gs.cameras->cols;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    checkCudaErrors(hipMalloc(&gs.cs, rows * cols * sizeof(hiprandState)));

    // int SHARED_SIZE_W_host;
#ifndef SHARED_HARDCODED
    int blocksize_w =
        gs.params->box_hsize + 1;  // +1 for the gradient computation
    int blocksize_h =
        gs.params->box_vsize + 1;  // +1 for the gradient computation
    WIN_RADIUS_W = (blocksize_w) / (2);
    WIN_RADIUS_H = (blocksize_h) / (2);

    int BLOCK_W = 32;
    int BLOCK_H = (BLOCK_W / 2);
    TILE_W = BLOCK_W;
    TILE_H = BLOCK_H * 2;
    SHARED_SIZE_W_m = (TILE_W + WIN_RADIUS_W * 2);
    SHARED_SIZE_H = (TILE_H + WIN_RADIUS_H * 2);
    SHARED_SIZE = (SHARED_SIZE_W_m * SHARED_SIZE_H);
    hipMemcpyToSymbol(HIP_SYMBOL(SHARED_SIZE_W), &SHARED_SIZE_W_m,
                       sizeof(SHARED_SIZE_W_m));
    // SHARED_SIZE_W_host = SHARED_SIZE_W_m;
#else
    // SHARED_SIZE_W_host = SHARED_SIZE;
#endif
    int shared_size_host = SHARED_SIZE;

    dim3 grid_size;
    grid_size.x = (cols + BLOCK_W - 1) / BLOCK_W;
    grid_size.y = ((rows / 2) + BLOCK_H - 1) / BLOCK_H;
    dim3 block_size;
    block_size.x = BLOCK_W;
    block_size.y = BLOCK_H;

    dim3 grid_size_initrand;
    grid_size_initrand.x = (cols + 16 - 1) / 16;
    grid_size_initrand.y = (rows + 16 - 1) / 16;
    dim3 block_size_initrand;
    block_size_initrand.x = 16;
    block_size_initrand.y = 16;

    size_t avail;
    size_t total;
    hipMemGetInfo(&avail, &total);
    size_t used = total - avail;
    int maxiter = gs.params->iterations;
    printf("Device memory used: %fMB\n", used / 1000000.0f);
    printf("Blocksize is %dx%d\n", gs.params->box_hsize, gs.params->box_vsize);

    printf("Number of iterations is %d\n", maxiter);
    gipuma_init_cu2<T><<<grid_size_initrand, block_size_initrand>>>(gs);

    hipDeviceSynchronize();
    hipEventRecord(start);
    // for (int it =0;it<gs.params.iterations; it++) {
    printf("Iteration ");
    for (int it = 0; it < 0; it++) {
        // for (int it = 0; it < maxiter; it++) {
        printf("%d ", it + 1);
        // spatial propagation of 4 closest neighbors (1px up/down/left/right)
        gipuma_black_spatialPropClose_cu<T>
            <<<grid_size, block_size, shared_size_host * sizeof(T)>>>(gs);
        hipDeviceSynchronize();

        // spatial propagation of 4 far away neighbors (5px up/down/left/right)
        gipuma_black_spatialPropFar_cu<T>
            <<<grid_size, block_size, shared_size_host * sizeof(T)>>>(gs);
        hipDeviceSynchronize();

        // plane refinement
        gipuma_black_lineRefine_cu<T>
            <<<grid_size, block_size, shared_size_host * sizeof(T)>>>(gs);
        hipDeviceSynchronize();

        // spatial propagation of 4 closest neighbors (1px up/down/left/right)
        gipuma_red_spatialPropClose_cu<T>
            <<<grid_size, block_size, shared_size_host * sizeof(T)>>>(gs);
        hipDeviceSynchronize();

        // spatial propagation of 4 far away neighbors (5px up/down/left/right)
        gipuma_red_spatialPropFar_cu<T>
            <<<grid_size, block_size, shared_size_host * sizeof(T)>>>(gs);
        hipDeviceSynchronize();

        // plane refinement
        gipuma_red_lineRefine_cu<T>
            <<<grid_size, block_size, shared_size_host * sizeof(T)>>>(gs);
        hipDeviceSynchronize();
    }
    printf("\n");
    printf("here?\n");
    // printf("Computing final disparity\n");
    // gipuma_compute_disp<<<grid_size_initrand, block_size_initrand>>>(gs);
    hipDeviceSynchronize();
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("\t\tTotal time needed for computation: %f seconds\n",
           milliseconds / 1000.f);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));

    // print results to file
    hipFree(&gs.cs);
}

int runcuda(GlobalState &gs) {
    gipuma<float>(gs);
    return 0;
}
